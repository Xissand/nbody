#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "molecules.h"
#include "units.h"
#include <cmath>
#include <cstdlib>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <string>

using namespace std;

#ifdef __INTELLISENSE__
void __syncthreads();
#endif

int step = 0;

float4* device_q;
float4* device_v;
float4* host_q;
float4* host_v;
float4* device_e;
float4* host_e;
Molecule* device_mol;
Molecule* host_mol;
float4 params;

__device__ void get_force(float4 qi, float4 qj, Molecule moli, float3& fi)
{
    float3 r;
    float coeff = 1;
    r.x = qi.x - qj.x;
    r.y = qi.y - qj.y;
    r.z = qi.z - qj.z;

    /*if (r.x == 0 && r.y == 0 && r.z == 0)
        return;*/

#ifdef PERIODIC_BOUNDARIES
    /*if (r.x > cell_size)
        r.x -= 2 * cell_size;
    if (r.x <= -cell_size)
        r.x += 2 * cell_size;
    if (r.y > cell_size)
        r.y -= 2 * cell_size;
    if (r.y <= -cell_size)
        r.y += 2 * cell_size;
    if (r.z > cell_size)
        r.z -= 2 * cell_size;
    if (r.z <= -cell_size)
        r.z += 2 * cell_size;*/

    r.x -= roundf(r.x / (2 * cell_size)) * (2 * cell_size);
    r.y -= roundf(r.y / (2 * cell_size)) * (2 * cell_size);
    r.z -= roundf(r.z / (2 * cell_size)) * (2 * cell_size);
#endif

    float r2 = r.x * r.x + r.y * r.y + r.z * r.z;

#ifdef LENNARD_JONES_POTENTIAL
    r2 = r2 / (moli.SIGMA * moli.SIGMA);
    float r4 = r2 * r2;
    float r6 = r4 * r2;
    float r8 = r4 * r4;

    float k = (-(0.5f) + (1.0f / r6)) * 12 / r8;
    coeff = 4 * moli.EPSILON;
#endif

#ifdef COULOMB_POTENTIAL
    if (r2 < 0.0001)
        r2 = 0.0001;

    float r_mod = sqrtf(r2);
    float r3 = r_mod * r_mod * r_mod;

    float k += 1.0f / r3;
#endif

    fi.x += r.x * coeff * k;
    fi.y += r.y * coeff * k;
    fi.z += r.z * coeff * k;
}

__device__ void get_virial(float4 qi, float4 qj, Molecule moli, float4& e)
{
    float3 r;
    float coeff = 1;
    r.x = qi.x - qj.x;
    r.y = qi.y - qj.y;
    r.z = qi.z - qj.z;

    /*if (r.x == 0 && r.y == 0 && r.z == 0)
        return;*/

#ifdef PERIODIC_BOUNDARIES
    /*if (r.x > cell_size)
        r.x -= 2 * cell_size;
    if (r.x <= -cell_size)
        r.x += 2 * cell_size;
    if (r.y > cell_size)
        r.y -= 2 * cell_size;
    if (r.y <= -cell_size)
        r.y += 2 * cell_size;
    if (r.z > cell_size)
        r.z -= 2 * cell_size;
    if (r.z <= -cell_size)
        r.z += 2 * cell_size;*/

    r.x -= roundf(r.x / (2 * cell_size)) * (2 * cell_size);
    r.y -= roundf(r.y / (2 * cell_size)) * (2 * cell_size);
    r.z -= roundf(r.z / (2 * cell_size)) * (2 * cell_size);
#endif

    float r2 = r.x * r.x + r.y * r.y + r.z * r.z;

#ifdef LENNARD_JONES_POTENTIAL
    r2 = r2 / (moli.SIGMA * moli.SIGMA);
    float r4 = r2 * r2;
    float r6 = r4 * r2;
    float r8 = r4 * r4;

    coeff = 4 * moli.SIGMA;
    float k = (-(0.5f) + (1.0f / r6)) * 12 / r8;
#endif

#ifdef COULOMB_POTENTIAL
    if (r2 < 0.0001)
        r2 = 0.0001;

    float r_mod = sqrtf(r2);
    float r3 = r_mod * r_mod * r_mod;

    float k += 1.0f / r3;
#endif

    float3 temp;
    temp.x = r.x * coeff * k;
    temp.y = r.y * coeff * k;
    temp.z = r.z * coeff * k;

    k = temp.x * r.x + temp.y * r.y + temp.z * r.z;
    e.w += k;
}

__device__ void get_potential(float4 qi, float4 qj, Molecule moli, float4& e)
{
    float p = 0;
    float3 r;
    float coeff = 1;
    r.x = qi.x - qj.x;
    r.y = qi.y - qj.y;
    r.z = qi.z - qj.z;

    /*if (r.x == 0 && r.y == 0 && r.z == 0)
        return;*/

#ifdef PERIODIC_BOUNDARIES
    /*if (r.x > cell_size)
        r.x -= 2 * cell_size;
    if (r.x <= -cell_size)
        r.x += 2 * cell_size;
    if (r.y > cell_size)
        r.y -= 2 * cell_size;
    if (r.y <= -cell_size)
        r.y += 2 * cell_size;
    if (r.z > cell_size)
        r.z -= 2 * cell_size;
    if (r.z <= -cell_size)
        r.z += 2 * cell_size;*/
    r.x -= roundf(r.x / (2 * cell_size)) * (2 * cell_size);
    r.y -= roundf(r.y / (2 * cell_size)) * (2 * cell_size);
    r.z -= roundf(r.z / (2 * cell_size)) * (2 * cell_size);
#endif

    float r2 = r.x * r.x + r.y * r.y + r.z * r.z;

#ifdef LENNARD_JONES_POTENTIAL
    r2 = r2 / (moli.SIGMA * moli.SIGMA);
    float r4 = r2 * r2;
    float r6 = r4 * r2;

    coeff = 4 * moli.EPSILON;

    p = (-(1.0f) + (1.0f / r6)) * coeff / r6;

    e.x += p / 2.0f;
    e.z += p / 2.0f;
#endif

#ifdef COULOMB_POTENTIAL
    if (r2 < 0.0001)
        r2 = 0.0001;

    float r_mod = sqrtf(r2);

    p += 1.0f / r_mod;
    e.x += p / 2;
    e.z += p / 2;
#endif
}

__device__ void get_kinetic(float4 v, Molecule mol, float4& e)
{
    float k = (v.x * v.x + v.y * v.y + v.z * v.z) / 2.0f;
    e.y += mol.M * k;
    e.z += mol.M * k;
}

__global__ void evolve(float4* d_q, float4* d_v, Molecule* d_mol, int N_BODIES, float dt,
                       float4* d_e, bool snap)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ float4 shared_q[BLOCK_SIZE];
    float4 q = d_q[j];
    float4 v = d_v[j];
    Molecule mol = d_mol[j];
    float4 e = {0.0f, 0.0f, 0.0f, 0.0f};
    float3 f = {0.0f, 0.0f, 0.0f};
    float3 a = {0.0f, 0.0f, 0.0f};
    float e_pot = 0;

    for (int i = 0; i < N_BODIES; i += BLOCK_SIZE)
    {
        shared_q[threadIdx.x] = d_q[i + threadIdx.x];
        __syncthreads();
        for (int k = 0; k < BLOCK_SIZE; k++)
        {
            if(k+i==j)
                continue;
            // if()
            get_force(q, shared_q[k], mol, f);
            if (snap)
            {
                get_potential(q, shared_q[k], mol, e);
                get_virial(q, shared_q[k], mol, e);
            }
        }
        __syncthreads();
    }
    if (snap)
        get_kinetic(v, mol, e);

    float m = mol.M;
    a.x = f.x / m;
    a.y = f.y / m;
    a.z = f.z / m;

    float a2 = a.x * a.x + a.y * a.y + a.z * a.z;
    /* if (snap)
         if (a2 != 0)
             printf("%f\n", sqrt(a2));*/

    v.x += a.x * dt;
    v.y += a.y * dt;
    v.z += a.z * dt;
    __syncthreads();
    // float e_kin = (v.x * v.x + v.y * v.y + v.z * v.z) / 2.0f;

    q.x += v.x * dt;
    q.y += v.y * dt;
    q.z += v.z * dt;

#ifdef PERIODIC_BOUNDARIES
    /* if (q.x > cell_size)
         q.x -= 2 * cell_size;
     if ((-q.x) > cell_size)
         q.x += 2 * cell_size;
     if (q.y > cell_size)
         q.y -= 2 * cell_size;
     if ((-q.y) > cell_size)
         q.y += 2 * cell_size;
     if (q.z > cell_size)
         q.z -= 2 * cell_size;
     if ((-q.z) > cell_size)
         q.z += 2 * cell_size;*/
#endif

    // e.x += e_pot / 2;
    // e.y += e_kin;
    // e.z += e_kin;

// Reflective boundaries
#ifdef REFLECTIVE_BOUNDARIES
    if (q.x > cell_size)
    {
        q.x -= 2 * (q.x - cell_size);
        v.x = -v.x;
    }
    if (q.x < -cell_size)
    {
        q.x += 2 * (-q.x - cell_size);
        v.x = -v.x;
    }
    if (q.y > cell_size)
    {
        q.y -= 2 * (q.y - cell_size);
        v.y = -v.y;
    }
    if (q.y < -cell_size)
    {
        q.y += 2 * (-q.y - cell_size);
        v.y = -v.y;
    }
    if (q.z > cell_size)
    {
        q.z -= 2 * (q.z - cell_size);
        v.z = -v.z;
    }
    if (q.z < -cell_size)
    {
        q.z += 2 * (-q.z - cell_size);
        v.z = -v.z;
    }
#endif

    __syncthreads();

    d_q[j] = q;
    d_v[j] = v;
    d_e[j] = e;
}

void generate()
{
    int limit = (int) truncf(cbrtf(N)) + 1;
    float grid_size = (2 * cell_size / limit);

    bool grid[limit][limit][limit];
    for (int i = 0; i < limit; i++)
        for (int j = 0; j < limit; j++)
            for (int k = 0; k < limit; k++)
                grid[i][j][k] = false;

    for (int n = 0; n < N; n++)
    {
        int grid_x = rand() % limit;
        int grid_y = rand() % limit;
        int grid_z = rand() % limit;

        if (grid[grid_x][grid_y][grid_z])
        {
            n--;
            continue;
        }

        float alpha = 0.1; // How much on grid particle are

        host_q[n].x =
            grid_x * grid_size + ((float) rand() / RAND_MAX) * alpha * grid_size - cell_size;
        host_q[n].y =
            grid_y * grid_size + ((float) rand() / RAND_MAX) * alpha * grid_size - cell_size;
        host_q[n].z =
            grid_z * grid_size + ((float) rand() / RAND_MAX) * alpha * grid_size - cell_size;
        host_q[n].w = 0;

        host_mol[n].set((MOLECULES) DEFAULT);

        // Rms for Maxwell disftibution is (3kT/m)**0.5, for rand is 1
        float velocity_coeff = sqrt((3 * K_B * T_INIT) / host_mol[n].M);

        host_v[n].x = velocity_coeff * 2 * ((float) rand() / RAND_MAX - 0.5);
        host_v[n].y = velocity_coeff * 2 * ((float) rand() / RAND_MAX - 0.5);
        host_v[n].z = velocity_coeff * 2 * ((float) rand() / RAND_MAX - 0.5);
        host_v[n].w = 1;

        // TODO: calculate starting energy
        host_e[n].x = 0; // Potential
        host_e[n].y = 0; // Kinetic
        host_e[n].z = 0; // Total
        host_e[n].w = 0; // Virial

        grid[grid_x][grid_y][grid_z] = true;
    }
}

void get_params(float4 e, float4& params)
{
    float P = 0, V = 0, T = 0;

    V = cell_size * cell_size * cell_size;
    T = e.y * 1 * 2 / 3;
    P = N * 1 * T / V + e.w / (3 * V);

    params = {P, V, T, 0};
}

void snapshot(ofstream& particles, ofstream& energy, ofstream& parameters)
{
    float E = 0, E_KIN = 0, E_POT = 0, VIRIAL = 0;
    if (SNAP_XYZ)
        hipMemcpy(host_q, device_q, sizeof(float4) * N, hipMemcpyDeviceToHost);
    hipMemcpy(host_e, device_e, sizeof(float4) * N, hipMemcpyDeviceToHost);

    if (SNAP_XYZ)
        particles << N << endl << endl;

    for (int i = 0; i < N; i++)
    {
        if (SNAP_XYZ)
        {
            particles << host_q[i].x << " ";
            particles << host_q[i].y << " ";
            particles << host_q[i].z << endl;
        }

        E += host_e[i].z;
        E_POT += host_e[i].x;
        E_KIN += host_e[i].y;
        VIRIAL += host_e[i].w;
    }

    energy << step << ",";
    energy << E_POT << ",";
    energy << E_KIN << ",";
    energy << E << ",";
    energy << VIRIAL << endl;

    get_params({E_POT, E_KIN, E, VIRIAL}, params);
    parameters << params.x << ",";
    parameters << params.y << ",";
    parameters << params.z << endl;

    cout << "Step: " << step << " ";
    cout << setprecision(15) << "Energy: " << E << " ";
    cout << "Temperature: " << params.z << endl;
}

void load_dump(string name)
{
    ifstream dump(name);

    for (int i = 0; i < N; i++)
    {
        dump >> host_q[i].x;
        dump >> host_q[i].y;
        dump >> host_q[i].z;

        dump >> host_v[i].x;
        dump >> host_v[i].y;
        dump >> host_v[i].z;
    }

    dump.close();
}

void create_dump(string name)
{
    ofstream dump(name);

    hipMemcpy(host_q, device_q, sizeof(float4) * N, hipMemcpyDeviceToHost);
    hipMemcpy(host_v, device_v, sizeof(float4) * N, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++)
    {
        dump << host_q[i].x << " ";
        dump << host_q[i].y << " ";
        dump << host_q[i].z << " ";

        dump << host_v[i].x << " ";
        dump << host_v[i].y << " ";
        dump << host_v[i].z << endl;
    }

    dump.close();
}

int main()
{
    hipMalloc(&device_q, sizeof(float4) * N);
    hipMalloc(&device_v, sizeof(float4) * N);
    hipMalloc(&device_e, sizeof(float4) * N);
    hipMalloc(&device_mol, sizeof(Molecule) * N);

    host_q = (float4*) malloc(sizeof(float4) * N);
    host_v = (float4*) malloc(sizeof(float4) * N);
    host_e = (float4*) malloc(sizeof(float4) * N);
    host_mol = (Molecule*) malloc(sizeof(Molecule) * N);

    generate();
    // load_dump("dump/dump.dat");

    ofstream particles("data/particles.xyz");
    ofstream energy("data/gpue.csv");
    energy << "t,Potential,Kinetic,Total,Virial" << endl;
    ofstream velocity("data/gpuv.csv");
    velocity << "vx,vy,vz,v" << endl;
    ofstream parameters("data/gpuparam.csv");
    parameters << "P,V,T" << endl;

    hipMemcpy(device_q, host_q, sizeof(float4) * N, hipMemcpyHostToDevice);
    hipMemcpy(device_v, host_v, sizeof(float4) * N, hipMemcpyHostToDevice);
    hipMemcpy(device_e, host_e, sizeof(float4) * N, hipMemcpyHostToDevice);
    hipMemcpy(device_mol, host_mol, sizeof(Molecule) * N, hipMemcpyHostToDevice);

    bool snap = false;
    for (step = 0; step < total_steps; step++)
    {
        if (step % snap_steps == 0)
            snap = true;
#ifndef __INTELLISENSE__
        evolve<<<N / BLOCK_SIZE, BLOCK_SIZE>>>(device_q, device_v, device_mol, N, dt, device_e,
                                               snap);
#endif
        if (step % snap_steps == 0)
            snapshot(particles, energy, parameters);
        snap = false;

        /*if(step == 100)
            create_dump("dump/dump.dat");*/
    }

    hipMemcpy(host_v, device_v, sizeof(float4) * N, hipMemcpyDeviceToHost);
    float v2 = 0;
    for (int i = 0; i < N; i++)
    {
        v2 = host_v[i].x * host_v[i].x + host_v[i].y * host_v[i].y + host_v[i].z * host_v[i].z;
        velocity << host_v[i].x << ",";
        velocity << host_v[i].y << ",";
        velocity << host_v[i].z << ",";
        velocity << v2 << endl;
    }

    particles.close();
    energy.close();
    velocity.close();
    parameters.close();

    hipFree(device_q);
    hipFree(device_v);
    hipFree(device_e);
    hipFree(device_mol);
    delete (host_q);
    delete (host_v);
    delete (host_e);
    delete (host_mol);

    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
