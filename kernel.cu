#include "hip/hip_runtime.h"
//#include "kernel.h"

#include <cmath>
#include <iostream>
#include <stdio.h>

#define PERIODIC_BOUNDARIES
//#define COULOMB_POTENTIAL
#define LENNARD_JONES_POTENTIAL

#define BLOCK_SIZE 128
const int cell_size = 8;
#define N 4096

float4* device_q;
float4* device_v;
float4* host_q;
float4* host_v;
float3* device_e;
float3* host_e;

__device__ void interract(float4 qi, float4 qj, float3& ai, float& p)
{
    float3 r;
    r.x = qi.x - qj.x;
    r.y = qi.y - qj.y;
    r.z = qi.z - qj.z;

    if (r.x == 0 && r.y == 0 && r.z == 0)
        return;

// Periodic boundary checks
#ifdef PERIODIC_BOUNDARIES
    if (r.x > cell_size)
        r.x -= 2 * cell_size;
    if (r.x <= -cell_size)
        r.x += 2 * cell_size;
    if (r.y > cell_size)
        r.y -= 2 * cell_size;
    if (r.y <= -cell_size)
        r.y += 2 * cell_size;
    if (r.z > cell_size)
        r.z -= 2 * cell_size;
    if (r.z <= -cell_size)
        r.z += 2 * cell_size;
#endif

    float r2 = r.x * r.x + r.y * r.y + r.z * r.z;

    float k = 0;
    // apparently powf can be more expensive, not sure

#ifdef LENNARD_JONES_POTENTIAL
    // p=r_12-r_6
    // f=-r_14+r_8

    float r4 = r2 * r2;
    float r6 = r4 * r2;
    float r8 = r4 * r4;

    k = (-(0.5f) + (1.0f / r6)) * 12 / r8;
    p += (-(1.0f) + (1.0f / r6)) * 1 / r6;
#endif

#ifdef COULOMB_POTENTIAL
    if (r2 < 0.0001)
        r2 = 0.0001;

    float r_mod = sqrtf(r2);
    float r3 = r_mod * r_mod * r_mod;

    k += 1.0f / r3;
    p += 1.0f / r_mod;
#endif

    ai.x += r.x * k;
    ai.y += r.y * k;
    ai.z += r.z * k;
    // printf("%f\n", r2);

    // printf("%f %f %f\n", ai.x, ai.y, ai.z);
}

__global__ void evolve(float4* d_q, float4* d_v, int N_BODIES, float dt, float3* d_e)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ float4 shared_q[BLOCK_SIZE];
    float4 q = d_q[j];
    float4 v = d_v[j];
    float3 e = {0.0f, 0.0f, 0.0f};
    float3 currA = {0.0f, 0.0f, 0.0f};
    float e_pot = 0;

    for (int i = 0; i < N_BODIES; i += BLOCK_SIZE)
    {
        shared_q[threadIdx.x] = d_q[i + threadIdx.x];
        __syncthreads();
        for (int k = 0; k < BLOCK_SIZE; k++)
        {
            interract(q, shared_q[k], currA, e_pot);
        }
        __syncthreads();
    }

    v.x += currA.x * dt;
    v.y += currA.y * dt;
    v.z += currA.z * dt;

    float e_kin = (v.x * v.x + v.y * v.y + v.z * v.z) / 2.0f;

    q.x += v.x * dt;
    q.y += v.y * dt;
    q.z += v.z * dt;

#ifdef PERIODIC_BOUNDARIES
    if (q.x > cell_size)
        q.x -= 2 * cell_size;
    if ((-q.x) > cell_size)
        q.x += 2 * cell_size;
    if (q.y > cell_size)
        q.y -= 2 * cell_size;
    if ((-q.y) > cell_size)
        q.y += 2 * cell_size;
    if (q.z > cell_size)
        q.z -= 2 * cell_size;
    if ((-q.z) > cell_size)
        q.z += 2 * cell_size;
#endif

    float currA2 = currA.x * currA.x + currA.y * currA.y + currA.z * currA.z;
    e.x += e_pot / 2;
    e.y += e_kin;
    e.z += e_pot / 2 + e_kin;

// Reflective boundaries
#ifdef REFLECTIVE_BOUNDARIES
    if (q.x > cell_size)
    {
        q.x -= 2 * (q.x - cell_size);
        v.x = -v.x;
    }
    if (q.x < -cell_size)
    {
        q.x += 2 * (-q.x - cell_size);
        v.x = -v.x;
    }
    if (q.y > cell_size)
    {
        q.y -= 2 * (q.y - cell_size);
        v.y = -v.y;
    }
    if (q.y < -cell_size)
    {
        q.y += 2 * (-q.y - cell_size);
        v.y = -v.y;
    }
    if (q.z > cell_size)
    {
        q.z -= 2 * (q.z - cell_size);
        v.z = -v.z;
    }
    if (q.z < -cell_size)
    {
        q.z += 2 * (-q.z - cell_size);
        v.z = -v.z;
    }
#endif

    __syncthreads();

    d_q[j] = q;
    d_v[j] = v;
    d_e[j] = e;
}

void generate()
{
    int grid_size = (int) truncf(2 * cell_size / (cbrtf(N)));
    int limit = (int) truncf(cbrtf(N));

    for (int i = 0; i < limit; i++)
        for (int j = 0; j < limit; j++)
            for (int k = 0; k < limit; k++)
            {
                int n = limit * limit * i + limit * j + k;
                host_q[n].x = i * grid_size + rand() % grid_size - cell_size;
                host_q[n].y = j * grid_size + rand() % grid_size - cell_size;
                host_q[n].z = k * grid_size + rand() % grid_size - cell_size;
                host_q[n].w = 0;

                host_v[n].x = (rand() % 100) / 10 - 5;
                host_v[n].y = (rand() % 100) / 10 - 5;
                host_v[n].z = (rand() % 100) / 10 - 5;
                host_v[n].w = 1;

                host_e[n].x = 0;
                host_e[n].y = 0;
                host_e[n].z = 0;
            }
    // TODO: implement generation for any number of particles
    // TODO: implement velocity generation based on temperature
}

int main()
{
    float dt = 0.001;
    int total_steps = 100000;
    int snap_steps = 5;
    float E = 0, E_KIN = 0, E_POT = 0;

    hipMalloc(&device_q, sizeof(float4) * N);
    hipMalloc(&device_v, sizeof(float4) * N);
    hipMalloc(&device_e, sizeof(float3) * N);

    host_q = (float4*) malloc(sizeof(float4) * N);
    host_v = (float4*) malloc(sizeof(float4) * N);
    host_e = (float3*) malloc(sizeof(float3) * N);

    generate();

    FILE* fp;
    fp = fopen("particles.xyz", "w");
    FILE* fp2;
    fp2 = fopen("gpue.csv", "w");
    fprintf(fp2, "t,Potential,Kinetic,Total\n");
    FILE* fp3;
    fp3 = fopen("gpuv.csv", "w");
    fprintf(fp3, "vx,vy,vz,v\n");

    hipMemcpy(device_q, host_q, sizeof(float4) * N, hipMemcpyHostToDevice);
    hipMemcpy(device_v, host_v, sizeof(float4) * N, hipMemcpyHostToDevice);
    hipMemcpy(device_e, host_e, sizeof(float3) * N, hipMemcpyHostToDevice);

    for (int step = 0; step < total_steps; step++)
    {
        evolve<<<N / BLOCK_SIZE, BLOCK_SIZE>>>(device_q, device_v, N, dt, device_e);
        if (step % snap_steps == 0)
        {
            E = E_POT = E_KIN = 0.0f;
            hipMemcpy(host_q, device_q, sizeof(float4) * N, hipMemcpyDeviceToHost);
            hipMemcpy(host_e, device_e, sizeof(float3) * N, hipMemcpyDeviceToHost);
            fprintf(fp, "%d\n\n", N);
            for (int i = 0; i < N; i++)
            {
                fprintf(fp, "%f %f %f\n", host_q[i].x, host_q[i].y, host_q[i].z);
                E += host_e[i].z;
                E_POT += host_e[i].x;
                E_KIN += host_e[i].y;
            }
            fprintf(fp2, "%d,%f,%f,%f\n", step, E_POT, E_KIN, E);
        }
    }

    hipMemcpy(host_v, device_v, sizeof(float4) * N, hipMemcpyDeviceToHost);
    float v2 = 0;
    for (int i = 0; i < N; i++)
    {
        v2 = host_v[i].x * host_v[i].x + host_v[i].y * host_v[i].y + host_v[i].z * host_v[i].z;
        fprintf(fp3, "%f,%f,%f,%f\n", host_v[i].x, host_v[i].y, host_v[i].z, v2);
    }

    fclose(fp);
    fclose(fp2);
    fclose(fp3);

    hipFree(device_q);
    hipFree(device_v);
    hipFree(device_e);
    delete (host_q);
    delete (host_v);
    delete (host_e);

    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
